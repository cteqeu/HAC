
#include "stdio.h"
#include "hip/hip_runtime.h"


__global__ void kernel()
{
  printf("Hello From CUDA Device\n");
}

int main()
{
  printf("In Host...\n");
  printf("Starting Kernel on device...\n");
  kernel<<<10,10>>>();
  hipDeviceSynchronize();
  return 0;
}
