
#include "stdio.h"
#include "hip/hip_runtime.h"


__global__ void kernel()
{
    printf("threadIdx.x : %d\n", threadIdx.x);

}

int main()
{
  printf("In Host...\n");
  printf("Starting Kernel on device...\n");
  kernel<<<1,10>>>();
  hipDeviceSynchronize();
  return 0;
}
