#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"



__global__ void gpu_testfunction(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x * blockDim.x+threadIdx.x;
	if (id < n)
	{
		a[id]=id;
		b[id]=id*id;
		c[id]=a[id]+b[id];
	}
}

__global__ void gpu_testfunction_uni_mem(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x * blockDim.x+threadIdx.x;
	if (id < n)
	{
		a[id]=id;
		b[id]=id*id;
		c[id]=a[id]+b[id];
	}
}

float *cpu_testfunction(float *a, float *b, int n)
{
	float *c;
    int i=0;
	size_t bytes = n*sizeof(float);
	c = (float*)malloc(bytes);
	
    for(i=0; i<n; i++){
        a[i]=i;
        b[i]=i*i;
    }
    
    for(i=0; i<n; i++){
        c[i] = a[i] + b[i];
    }
	return c;
}

int main( int argc, char *argv[] )
{
    
    int n = 100000;
    
    float *a;
    float *b;
    float *c;
    
    float *d_a;
    float *d_b;
    float *d_c;
	
	float *a_um;
	float *b_um;
	float *c_um;
	
    size_t bytes = n*sizeof(float);
    
    a = (float*)malloc(bytes);
    b = (float*)malloc(bytes);
    c = (float*)malloc(bytes);
    
	hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
	
	hipMallocManaged(&a_um, bytes);
	hipMallocManaged(&b_um, bytes);
	hipMallocManaged(&c_um, bytes);
	
	c=cpu_testfunction(a,b,n);

    float control = 0;
	int i=0;
    for(i=0; i<n; i++) {
        control += c[i];
    }
	
    printf("cpu control: \t\t%f\n", control);
    
	hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);
	
	int blockSize, gridSize;

    blockSize = 1024;

    gridSize = (int)ceil((float)n/blockSize);
 
    gpu_testfunction<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost );

    control = 0;
    i=0;
    for(i=0; i<n; i++) {
        control += c[i];
    }
    printf("gpu control: \t\t%f\n", control);
	
	gpu_testfunction_uni_mem<<<gridSize, blockSize>>>(a_um, b_um, c_um, n);
	hipDeviceSynchronize();

    control = 0;
    i=0;
    for(i=0; i<n; i++) {
        control += c[i];
    }
    printf("gpu um control: \t%f\n", control);

    free(a);
    free(b);
    free(c);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	hipFree(a_um);
	hipFree(b_um);
	hipFree(c_um);
    
    return 0;
}