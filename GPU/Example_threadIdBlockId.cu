#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define n 10000000

__global__ void vecAdd(float *a, float *b, float *c, int L)
{

    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < L)
        c[id] = a[id] + b[id];
}

int main()
{
    float *h_a;
    float *h_b;
    float *h_c;
 
    float *d_a;
    float *d_b;
    float *d_c;
 
    size_t bytes = n*sizeof(float);

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    for( i = 0; i < n; i++ ) {
        h_a[i] = (float)i;
        h_b[i] = (float)i;
    }

    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
    blockSize = 1024;
    gridSize = (int)ceil((float)n/blockSize);
 
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
	printf("h_c[i] = %f\r\n",h_c[n-1]);
	
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}