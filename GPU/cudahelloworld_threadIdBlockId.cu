#include <stdio.h>
#include <stdlib.h> 
#include "hip/hip_runtime.h"


__global__ void print_from_gpu(void) {
	printf("Thread [%d,%d] From device\r\n", threadIdx.x,blockIdx.x); 
}

int main(void) { 
	printf("Print from Host!\r\n"); 
	print_from_gpu<<<2,2>>>();
	hipDeviceSynchronize();
return 0; 
}