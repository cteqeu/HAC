// Working with CUDA Runtime API
// Information :https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html

#include "stdio.h"
#include "hip/hip_runtime.h"


int main()
{
	int deviceIdx = 0;
	hipSetDevice(deviceIdx);
	
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceIdx);
	printf("GPU is %s, index set is %d\n",deviceProp.name, deviceIdx);
  return 0;
}